#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <string>
#include <iostream>
#include <cstdio>
#include <ctime>
#include <fstream>
#include <Windows.h>


using namespace std;

const unsigned PRIME_BASE = 251;

//max size 4 byte prime
const long unsigned PRIME_MOD = 4294967291;
// max substring length 8555711

unsigned const BLOCKS = 40;
unsigned const THREADS = 256;
//maximum size of chunk of data to process at a time
unsigned long long const CHUNK_SIZE = 1000000000;//1GB

long long rabinKarpCPU(char* string, char* substring, unsigned long long fileSize, bool pos);
unsigned int rollingHash(char* string);
__global__ void rabinKarpKernel(unsigned long long stringLenght, char* string, unsigned int substringLenght, char* substring, unsigned int substringHash, unsigned int power, unsigned int* matchesCount, bool pos);

int main(int argc, char** argv)
{
	if (argc == 1 || strcmp(argv[1], "-help") == 0 || strcmp(argv[1], "help") == 0) {
		printf("Substring search in file help\n");
		printf("First argument is -g for GPU calculation or -c for CPU calculation\n");
		printf("Second argument is path to the desired file to be searched\n");
		printf("Third argument is the substring to be searched for\n");
		printf("Fourth argument is either -count for count of matches or -pos for individual positions of the matches; -pos does not work properly fo GPU calculation\n");
		printf("IF using the GPU calculation, the input file has to be at least 15000 times larger that the length of the substring\n");
		return -1;
	}
	if (argc < 5) {
		printf("Not enough arguments\n");
		return -1;
	}
	if (argc > 5) {
		printf("Too many arguments\n");
		return -1;
	}

	//opening and mapping a file to memory

	//open existing file in read mode and get its handle
	HANDLE hFile = CreateFile(argv[2], GENERIC_READ, 0, NULL, OPEN_EXISTING, FILE_ATTRIBUTE_NORMAL, NULL);
	if (hFile == NULL)
	{
		printf("hFile is NULL: last error: %d\n"), GetLastError();
		return (2);
	}

	//get the file size
	LARGE_INTEGER fs;
	GetFileSizeEx(hFile, &fs);
	unsigned long long fileSize = fs.QuadPart;

	//map entire file to to memory in readonly mode 
	HANDLE hMapFile = CreateFileMapping(hFile, NULL, PAGE_READONLY, 0, 0, NULL);
	if (hMapFile == NULL)
	{
		printf("hMapFile is NULL: last error: %d\n"), GetLastError();
		return (2);
	}

	//create a view of the entire memory mapped file and get a pointer to the start adresses
	LPVOID lpMapAddress = MapViewOfFile(hMapFile, FILE_MAP_READ, 0, 0, 0);
	char* data = (char *)lpMapAddress;

	if (lpMapAddress == NULL)
	{
		printf("lpMapAddress is NULL: last error: %d\n"), GetLastError();
		return (2);
	}

	//start timing
	clock_t start;
	double duration;
	start = clock();

	char* stringToFind = argv[3];
	int stringToFindLen = strlen(stringToFind);
	long totMatches = 0;

	bool pos;
	if (strcmp(argv[4], "-pos") == 0) {
		pos = 1;
	}
	else {
		pos = 0;
	}

	//cpu calculation
	if (strcmp(argv[1], "-c") == 0) {
		totMatches = rabinKarpCPU(data, stringToFind, fileSize, pos);
	}
	else if (strcmp(argv[1], "-g") == 0) {
	//GPU calculation
		unsigned int matchesCount = 0;
		unsigned int* d_matchesCount;
		char* d_substring;
		char* d_string;
		unsigned long long alreadyDone = 0;
		unsigned long long currentChunkSize = 0;
		bool dataLeft = 0;

		//allocate memory on device
		if (hipSuccess != hipMalloc(&d_matchesCount, sizeof(unsigned int))) {
			printf("Cuda memory allocation failed\n");
			return -1;
		}
		if (hipSuccess != hipMalloc(&d_substring, stringToFindLen)) {
			printf("Cuda memory allocation failed\n");
			return -1;
		}
		if (hipSuccess != hipMalloc(&d_string, CHUNK_SIZE)) {
			printf("Cuda memory allocation failed\n");
			return -1;
		}

		//main loop
		//separate the memory mapped file into chunks that cen be processed at one time
		while (alreadyDone < fileSize)
		{

			//decide what is the current chunk size and if there is still data left to process
			if (fileSize <= CHUNK_SIZE) {
				currentChunkSize = fileSize;
				dataLeft = 0;
				//printf("setting chunk size to filesize\n");
			}
			else if (fileSize <= (alreadyDone + (CHUNK_SIZE - (stringToFindLen - 1)))) {
				currentChunkSize = fileSize - alreadyDone;
				dataLeft = 0;
				//printf("setting chunk size to filesize - alredy done\n");
			}
			else {
				currentChunkSize = CHUNK_SIZE;
				dataLeft = 1;
				//printf("setting chunk size to CHUNK_SIZE, alredy done:%llu\n", alreadyDone);
			}


			//clear the memory and copy data to the device
			hipMemset(d_matchesCount, 0, sizeof(unsigned int));
			hipMemset(d_string, 0, currentChunkSize);
			hipMemcpy(d_substring, stringToFind, stringToFindLen, hipMemcpyHostToDevice);
			hipMemcpy(d_string, data, currentChunkSize, hipMemcpyHostToDevice);

			//if there is data left move pointer to start of next chunk and increment the done counter, otherwise only increment the done counter
			if (dataLeft) {
				data += currentChunkSize - (stringToFindLen - 1);
				alreadyDone += currentChunkSize - (stringToFindLen - 1);
			}
			else {
				alreadyDone += currentChunkSize;
			}

			unsigned int power = 1;
			for (int i = 0; i < stringToFindLen; i++)
				power = (power * PRIME_BASE) % PRIME_MOD;

			//launch the kernel
			rabinKarpKernel << <BLOCKS, THREADS >> > (currentChunkSize, d_string, stringToFindLen, d_substring, rollingHash(stringToFind), power, d_matchesCount, pos);
			hipDeviceSynchronize(); // technically not required 
			//copy data back to host
			hipMemcpy(&matchesCount, d_matchesCount, sizeof(unsigned int), hipMemcpyDeviceToHost);
			totMatches += matchesCount;
		}
		hipFree(d_string);
		hipFree(d_matchesCount);
		hipFree(d_substring);
	}
	else {
		printf("First argument invalid, use -c for CPU search or -g for GPU search\n");
		return -1;
	}

	duration = (clock() - start) / (double)CLOCKS_PER_SEC;

	if (strcmp(argv[4], "-count") == 0) {
		printf("%d\n", totMatches);
	}
	cout << "Program runtime: " << duration << '\n';


	bool bFlag;
	bFlag = UnmapViewOfFile(lpMapAddress);
	bFlag = CloseHandle(hMapFile); // close the file mapping object

	if (!bFlag)
	{
		printf("\nError %ld occurred closing the mapping object!"), GetLastError();
	}

	bFlag = CloseHandle(hFile);   // close the file itself

	if (!bFlag)
	{
		printf("\nError %ld occurred closing the file!"), GetLastError();
	}

	return 0;
}

unsigned int rollingHash(char* string)
{
	unsigned int ret = 0;
	for (int i = 0; i < strlen(string); i++)
	{
		ret = ret*PRIME_BASE + string[i];
		ret %= PRIME_MOD; //don't overflow
	}
	return ret;
}


__global__ void rabinKarpKernel(unsigned long long stringLenght, char* string, unsigned int substringLenght, char* substring, unsigned int substringHash, unsigned int power, unsigned int* matchesCount, bool pos) {

	int id = blockIdx.x * blockDim.x + threadIdx.x; // id of a thread
	unsigned long chunkSize = (stringLenght + (BLOCKS*THREADS) - 1) / (BLOCKS*THREADS); // amount of data to process by a single thread
	unsigned int rollingHash = 0;
	int mm = 0;

	if (id < stringLenght) {
		for (unsigned long long i = chunkSize*id; i < chunkSize*(id + 1) + substringLenght - 1; i++)
		{
			if (i < stringLenght) {
				//add the last letter
				rollingHash = rollingHash*PRIME_BASE + string[i];
				rollingHash %= PRIME_MOD;

				//remove the first character, if needed
				if (i >= chunkSize*id + substringLenght)
				{
					rollingHash -= power * string[i - substringLenght] % PRIME_MOD;
				}

				//match?
				if (i >= substringLenght - 1 && rollingHash == substringHash) {
					bool match = 1;
					for (int j = 0; j < substringLenght; j++)
					{
						if (string[i - j] != substring[substringLenght - j - 1]) {
							match = 0;
						}
					}
					if (match) {
						mm = atomicAdd(matchesCount, 1); // atomicly incerement a found string counter
						if (pos) {
							printf("Match %d found by thread %d at: %llu\n",mm ,id , i - substringLenght);// WHY DO YOU HATE ME
						}
					}
				}
			}
		}
	}
}

long long rabinKarpCPU(char* string, char* substring, unsigned long long fileSize, bool positions)
{
	unsigned int substringHash = rollingHash(substring);
	unsigned int rollingHash = 0;
	long long count = 0;

	unsigned long long stringLength = fileSize;
	unsigned long long substringLength = strlen(substring);

	//this is the max power for hash, meaning we can access the topmost hashed character just by multiplying with this power
	unsigned int power = 1;
	for (int i = 0; i < substringLength; i++)
		power = (power * PRIME_BASE) % PRIME_MOD;

	for (unsigned long long i = 0; i < stringLength; i++)
	{
		//add the last letter
		rollingHash = rollingHash*PRIME_BASE + string[i];
		rollingHash %= PRIME_MOD;

		//remove the first character, if needed
		if (i >= substringLength)
		{
			rollingHash -= power * string[i - substringLength] % PRIME_MOD;
		}

		//match?
		if (i >= substringLength - 1 && rollingHash == substringHash) {

			bool match = 1;
			for (int j = 0; j < substringLength; j++)
			{
				if (string[i - j] != substring[substringLength - j - 1]) {
					match = 0;
				}
			}
			if (match) {
				if (positions) {
					printf("Match found at postition: %llu\n", i - substringLength);
				}
				count++;
			}
		}
	}
	return count;
}


